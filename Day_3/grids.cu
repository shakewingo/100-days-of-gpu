// A program demonstrating the use of CUDA grids and blocks
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

dim3 dimGrid(32, 1, 1); 
dim3 dimBlock(128, 1, 1);

__global__ void kernel(void){}

int main(void) {
  kernel<<<dimGrid, dimBlock>>>();  // Define how many blocks and threads will be parallelly executed
  hipDeviceSynchronize(); // Ensure the kernel has finished executing before exiting
  int totalThreads = dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x * dimBlock.y * dimBlock.z;
  cout << "Total number of threads: " << totalThreads << endl;  // it uses the standard C++ cout stream from the <iostream> header
  return 0;
} 